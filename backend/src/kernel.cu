#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx];
    }
}

# called in backend.cpp
extern "C" void launch_kernel(const float* input, float* output, size_t size) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;    

    float* deviceInput = nullptr;
    float * deviceOutput = nullptr;
    size_t bytes = size * sizeof(float);

    hipMalloc((void**)&deviceInput, bytes);
    hipMalloc((void**)&deviceOutput, bytes);

    hipMemcpy(deviceInput, input, bytes, hipMemcpyHostToDevice);

    kernel<<<gridSize,blockSize>>>(deviceInput, deviceOutput, size);

    hipMemcpy(output, deviceOutput, bytes, hipMemcpyDeviceToHost);

    hipFree(deviceInput);
    hipFree(deviceOutput); 
     
}
