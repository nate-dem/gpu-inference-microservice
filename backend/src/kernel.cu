#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx];
    }
}

extern "C" void launch_kernel(const float* input, float* output, size_t size) {
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;    

    float* d_in = nullptr;
    float* d_out = nullptr;
    size_t bytes = size * sizeof(float);

    hipMalloc((void**)&d_in, bytes);
    hipMalloc((void**)&d_out, bytes);

    hipMemcpy(d_in, input, bytes, hipMemcpyHostToDevice);

    kernel<<<gridSize,blockSize>>>(d_in, d_out, size);

    hipMemcpy(output, d_out, bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);      
}
